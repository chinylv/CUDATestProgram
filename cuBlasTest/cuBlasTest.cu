//Example A.2. Application Using C and cuBLAS: 0-based indexing
//----------------------------------------------------------- 

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include "hipblas.h" 
#define M 6 
#define N 5 
#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 

static __inline__ void modify (float *m, int ldm, int n, int p, int q, float alpha, float beta){ 
cublasSscal (n-p, alpha, &m[IDX2C(p,q,ldm)], ldm); 
cublasSscal (ldm-p, beta, &m[IDX2C(p,q,ldm)], 1); 
} 

int main (void){ 
int i, j; 
hipblasStatus_t stat; 
float* devPtrA; 
float* a = 0; 
a = (float *)malloc (M * N * sizeof (*a)); 
if (!a) { 
	printf ("host memory allocation failed"); 
	return EXIT_FAILURE; 
} 

for (j = 0; j < N; j++) { 
	for (i = 0; i < M; i++) { 
		a[IDX2C(i,j,M)] = (float)(i * M + j + 1); 
	}
} 

cublasInit(); 

stat = cublasAlloc (M*N, sizeof(*a), (void**)&devPtrA); 

if (stat != HIPBLAS_STATUS_SUCCESS) { 
	printf ("device memory allocation failed"); 
	cublasShutdown(); 
	return EXIT_FAILURE; 
} 

stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M); 

if (stat != HIPBLAS_STATUS_SUCCESS) { 
	printf ("data download failed"); 
	cublasFree (devPtrA); 
	cublasShutdown(); 
	return EXIT_FAILURE; 
} 

modify (devPtrA, M, N, 1, 2, 16.0f, 12.0f); 
stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M); 

if (stat != HIPBLAS_STATUS_SUCCESS) { 
	printf ("data upload failed"); 
	cublasFree (devPtrA); 
	cublasShutdown(); 
	return EXIT_FAILURE; 
} 

cublasFree (devPtrA);
cublasShutdown(); 
for (j = 0; j < N; j++) { 
	for (i = 0; i < M; i++) { 
		printf ("%7.0f", a[IDX2C(i,j,M)]); 
	}
	printf ("\n");
} 

free(a);

return EXIT_SUCCESS;
}
