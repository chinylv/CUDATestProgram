
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#define M 16
#define N 128
#define K 512


int main(){

  float *d_a, *d_b, *d_c;
  const float alpha = 1.0f;
  const float beta = 0.0f;

  hipMalloc(&d_b, K*M*sizeof(float));
  hipMalloc(&d_c, N*M*sizeof(float));
  hipMalloc(&d_a, N*K*sizeof(float));

  hipblasHandle_t my_handle;
  hipblasStatus_t my_status = hipblasCreate(&my_handle);
  if (my_status != HIPBLAS_STATUS_SUCCESS) {printf("handle failure %d\n", (int)my_status); return 1;}
  hipMemset(d_a, 0, N*K*sizeof(float));
  hipMemset(d_b, 0, K*M*sizeof(float));

  my_status = hipblasSgemm(my_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_b, M, d_a, K, &beta, d_c, M);
  if (my_status != HIPBLAS_STATUS_SUCCESS) {printf("Sgemm failure %d\n", (int)my_status); return 1;}
  printf("Success\n");
  return 0;
}
